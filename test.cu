#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>


const int N = 8; 
const int perthreads = 8;
const int blockspergrid =std::min(32, (N + perthreads - 1) / perthreads);

__global__ void addKernel(int *a,int *b, int *c)
{
	__shared__ int cache[perthreads];
	int index = threadIdx.x + blockIdx.x * blockDim.x ;
	//	printf("index=%d\n",index);
	int tmp(0);
	int cacheindex(threadIdx.x);

	while (index < N) {
		tmp += a[index] * b[index];
		index += blockDim.x * gridDim.x;
	}
	cache[cacheindex] = tmp;
		printf("tmp=%d cacheindex=%d\n",tmp,cacheindex);
	__syncthreads();// 每个block中thread数量大于cudacore才需要？
	int fg = perthreads / 2;
	while (fg > 0) {
		if (cacheindex < fg) {
			cache[cacheindex] += cache[cacheindex + fg];
			__syncthreads();
		}
			fg /= 2;
			printf("cache[0]=%d,fg=%d",cache[0],fg);
	}
		printf("end while");
		printf("cache[0]=%d",cache[0]);
		printf("cache[1]=%d",cache[1]);
	if (cacheindex == 0){
		c[blockIdx.x] = cache[cacheindex];
		printf("blockIdx=%d\n",blockIdx.x);
	}
}

int main()
{
	int a[N],b[N],c[blockspergrid];
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i;
	}
	std::cout << "11="  << std::endl;

	int* dev_a(0), * dev_b(0), * dev_c(0);
	hipMalloc((void**) &dev_a,N * sizeof(int));
	hipMalloc((void**) &dev_b,N * sizeof(int));
	hipMalloc((void**) &dev_c,blockspergrid * sizeof(int));
	std::cout << "22="  << std::endl;
	
	hipMemcpy(dev_a, a,N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b,N * sizeof(int), hipMemcpyHostToDevice);

	addKernel << <blockspergrid, perthreads>> > (dev_a, dev_b, dev_c);
	std::cout << "33="  << std::endl;
		std::cout << "blockspergird=" << blockspergrid<< std::endl;
	hipMemcpy(c, dev_c, blockspergrid * sizeof(int),hipMemcpyDeviceToHost);
	int result(0);
	for (int i = 0; i < blockspergrid; i++) {
		std::cout << "result=" << result << std::endl;
		result += c[i];
	}
	std::cout << "result=" << result << std::endl;
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
